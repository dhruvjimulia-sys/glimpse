#include "hip/hip_runtime.h"
#include "pe.h"
 
__device__ __host__ bool getBitAt(uint8_t pixel_value, size_t bit_num) {
    if (bit_num >= 8) {
        printf("PD called more times than number of bits in image");
        return 0;
    } else {
        return (pixel_value & (1 << bit_num)) >> bit_num; 
    }
}

__device__ bool getNeighbourValue(
    bool* neighbour_shared_values,
    size_t neighbour_index,
    size_t num_shared_neighbours,
    size_t shared_neighbour_value
) {
    return neighbour_shared_values[neighbour_index * num_shared_neighbours + shared_neighbour_value - 1];
}

__device__ bool getInstructionInputValue(
    InputC inputc,
    bool* memory,
    uint8_t* image,
    size_t pd_bit,
    bool* pd_increment,
    int64_t x,
    int64_t y,
    size_t image_x_dim,
    size_t image_y_dim,
    size_t image_size,
    size_t offset,
    bool* neighbour_shared_values,
    size_t num_shared_neighbours,
    size_t shared_neighbour_value,
    bool use_shared_memory,
    bool* neighbour_shared_values_cache
) {
    bool input_value = false;
    switch (inputc.input.inputKind) {
        case InputKind::Address: input_value = memory[offset * MEMORY_SIZE_IN_BITS + inputc.input.address]; break;
        case InputKind::ZeroValue: input_value = false; break;
        case InputKind::PD:
            input_value = getBitAt(image[offset], pd_bit);
            *pd_increment = true;
            break;
        case InputKind::Up:
            if (y - 1 >= 0) {
                int64_t up_index = offset - image_x_dim;
                input_value = getNeighbourValue(
                    neighbour_shared_values,
                    up_index,
                    num_shared_neighbours,
                    shared_neighbour_value
                );
            } else {
                input_value = false;
            }
            break;
        case InputKind::Down:
            if (y + 1 < image_y_dim) {
                int64_t down_index = offset + image_x_dim;
                input_value = getNeighbourValue(
                    neighbour_shared_values,
                    down_index,
                    num_shared_neighbours,
                    shared_neighbour_value
                );
            } else {
                input_value = false;
            }
            break;
        case InputKind::Right:
            if (x + 1 < image_x_dim) {
                int64_t right_index = offset + 1;
                input_value = getNeighbourValue(
                    neighbour_shared_values,
                    right_index,
                    num_shared_neighbours,
                    shared_neighbour_value
                );
            } else {
                input_value = false;
            }
            break;
        case InputKind::Left:
            if (x - 1 >= 0) {
                int64_t left_index = offset - 1;
                input_value = getNeighbourValue(
                    neighbour_shared_values,
                    left_index,
                    num_shared_neighbours,
                    shared_neighbour_value
                );
            } else {
                input_value = false;
            }
            break;
        default:
            break;
    }
    return (inputc.negated) ? !input_value : input_value;
}

__global__ void processingElemKernel(
    size_t num_instructions,
    uint8_t* image,
    bool* neighbour_shared_values,
    bool* external_values,
    size_t image_size,
    size_t image_x_dim,
    size_t image_y_dim,
    size_t num_outputs,
    size_t num_shared_neighbours,
    size_t* debug_output,
    size_t num_debug_outputs,
    size_t vliw_width,
    bool use_shared_memory,
    bool is_pipelining,
    bool* local_memory_values,
    bool* carry_register_values,
    bool* result_values
) {
    size_t x = threadIdx.x + blockIdx.x * blockDim.x;
    size_t y = threadIdx.y + blockIdx.y * blockDim.y;
    size_t offset = x + y * blockDim.x * gridDim.x;
    // Note: PIPELINE_WIDTH
    const size_t PIPELINE_WIDTH = 3;
    
    if (offset < image_size) {
        cg::grid_group grid = cg::this_grid();
        
        // image_x, image_y in image space
        // x, y in thread/block space
        __shared__ bool neighbour_shared_values_cache[NUM_THREADS_PER_BLOCK_PER_DIM][NUM_THREADS_PER_BLOCK_PER_DIM];
        // if (use_shared_memory) {
        //     neighbour_shared_values_cache[threadIdx.y][threadIdx.x] = false;
        //     __syncthreads();
        // }
        size_t pd_bit = 0;
        bool pd_increment = false;
        size_t output_number = 0;

        // shared_neighbour_value is the index of the shared neighbour value
        bool shared_neighbour_value_increment = false;
        size_t shared_neighbour_value = 0;

        bool output_number_increment = false;

        bool contains_neighbour_sharing = false;

        for (size_t i = 0; (i < num_instructions && !is_pipelining) || (i < num_instructions + PIPELINE_WIDTH - 1 && is_pipelining); i++) {
            size_t offset = x + y * blockDim.x * gridDim.x;
            size_t image_x = offset % image_x_dim;
            size_t image_y = offset / image_x_dim;
            while (offset < image_size) {
                if (i < num_instructions) {
                    for (size_t j = 0; j < vliw_width; j++) { 
                        const Instruction instruction = ((Instruction *) dev_instructions)[i * vliw_width + j];
                        if (instruction.isNop) {
                            continue;
                        }
                        bool carryval = false;
                        switch (instruction.carry) {
                            case Carry::CR: carryval = carry_register_values[offset * vliw_width + j]; break;
                            case Carry::One: carryval = true; break;
                            case Carry::Zero: carryval = false; break;
                        }
                        bool input_one = getInstructionInputValue(
                            instruction.input1,
                            local_memory_values,
                            image,
                            pd_bit,
                            &pd_increment,
                            image_x,
                            image_y,
                            image_x_dim,
                            image_y_dim,
                            image_size,
                            offset,
                            neighbour_shared_values,
                            num_shared_neighbours,
                            shared_neighbour_value,
                            use_shared_memory,
                            (bool *) neighbour_shared_values_cache
                        );
                        bool input_two = getInstructionInputValue(
                            instruction.input2,
                            local_memory_values,
                            image,
                            pd_bit,
                            &pd_increment,
                            image_x,
                            image_y,
                            image_x_dim,
                            image_y_dim,
                            image_size,
                            offset,
                            neighbour_shared_values,
                            num_shared_neighbours,
                            shared_neighbour_value,
                            use_shared_memory,
                            (bool *) neighbour_shared_values_cache
                        );

                        // printf("offset: %lu, instruction: %lu, input_one: %d, carryval: %d, input_two: %d\n", offset, i, input_one, carryval, input_two);
                        
                        // debug_output value = 0 if nop
                        // debug_output[((offset * num_instructions + i) * vliw_width + j) * num_debug_outputs] = input_one;
                        // debug_output[((offset * num_instructions + i) * vliw_width + j) * num_debug_outputs + 1] = input_two;
                        // debug_output[((offset * num_instructions + i) * vliw_width + j) * num_debug_outputs + 2] = carryval;

                        const bool sum = (input_one != input_two) != carryval;
                        const bool carry = (carryval && (input_one != input_two)) || (input_one && input_two);

                        // Assuming can only be two values
                        result_values[(offset * vliw_width + j) * PIPELINE_WIDTH + (i % PIPELINE_WIDTH)] = (instruction.resultType.value == 's') ? sum : carry;

                        // Interesting choice...
                        if (instruction.carry == Carry::CR) {
                            carry_register_values[offset * vliw_width + j] = carry;
                        }
                    }
                }

                if (!is_pipelining || (is_pipelining && i >= PIPELINE_WIDTH - 1)) {
                    for (size_t j = 0; j < vliw_width; j++) {
                        const Instruction instruction = 
                        !is_pipelining ?
                        ((Instruction *) dev_instructions)[i * vliw_width + j] :
                        ((Instruction *) dev_instructions)[(i - PIPELINE_WIDTH + 1) * vliw_width + j];
                        if (instruction.isNop) {
                            continue;
                        }
                        size_t resultvalue = 
                        !is_pipelining ?
                        result_values[(offset * vliw_width + j) * PIPELINE_WIDTH + (i % PIPELINE_WIDTH)] :
                        result_values[(offset * vliw_width + j) * PIPELINE_WIDTH + ((i - PIPELINE_WIDTH + 1) % PIPELINE_WIDTH)];
                        switch (instruction.result.resultKind) {
                            case ResultKind::Address:
                                local_memory_values[offset * MEMORY_SIZE_IN_BITS + instruction.result.address] = resultvalue;
                                break;
                            case ResultKind::Neighbour:
                                neighbour_shared_values[offset * num_shared_neighbours + shared_neighbour_value] = resultvalue;
                                shared_neighbour_value_increment = true;
                                contains_neighbour_sharing = true;
                                break;
                            case ResultKind::External:
                                external_values[num_outputs * offset + output_number] = resultvalue;
                                output_number_increment = true;
                                break;
                        }
                    }
                }
                offset += blockDim.x * gridDim.x * blockDim.y * gridDim.y;
                image_x = offset % image_x_dim;
                image_y = offset / image_x_dim;
            }

            if (pd_increment) {
                pd_bit++;
            }
            pd_increment = false;
            if (shared_neighbour_value_increment) {
                shared_neighbour_value++;
            }
            shared_neighbour_value_increment = false;
            if (contains_neighbour_sharing) {
                grid.sync();
            }
            contains_neighbour_sharing = false;
            if (output_number_increment) {
                output_number++;
            }
            output_number_increment = false;
        }
    }
};
