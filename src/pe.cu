#include "hip/hip_runtime.h"
#include "pe.h"

__device__ bool getBitAt(uint8_t pixel_value, size_t bit_num) {
    if (bit_num >= 8) {
        printf("PD called more times than number of bits in image");
        return 0;
    } else {
        return pixel_value & (1 << bit_num); 
    }
}

// TODO Inline function
__device__ void waitUntilAvailable(
    bool* neighbour_shared_values,
    volatile size_t* neighbour_program_counter,
    size_t neighbour_pc,
    int64_t index,
    size_t image_size
) {
    while (neighbour_program_counter[index] != neighbour_pc);
}

__device__ bool getInstructionInputValue(
    InputC inputc,
    bool *memory,
    uint8_t* image,
    size_t pd_bit,
    int64_t x,
    int64_t y,
    size_t image_x_dim,
    size_t image_y_dim,
    size_t image_size,
    size_t offset,
    size_t* neighbour_program_counter,
    bool* neighbour_shared_values,
    size_t neighbour_update_pc
) {
    bool input_value = false;
    switch (inputc.input.inputKind) {
        case InputKind::Address: input_value = memory[inputc.input.address]; break;
        case InputKind::ZeroValue: input_value = false; break;
        case InputKind::PD: input_value = getBitAt(image[offset], pd_bit); pd_bit++; break;
        case InputKind::Up:
            if (y - 1 >= 0) {
                int64_t up_index = offset - image_x_dim;
                waitUntilAvailable(neighbour_shared_values, neighbour_program_counter, neighbour_update_pc, up_index, image_size);
                input_value = neighbour_shared_values[up_index];
            } else {
                input_value = false;
            }
            break;
        case InputKind::Down:
            if (y + 1 < image_y_dim) {
                int64_t down_index = offset + image_x_dim;
                waitUntilAvailable(neighbour_shared_values, neighbour_program_counter, neighbour_update_pc, down_index, image_size);
                input_value = neighbour_shared_values[down_index];
            } else {
                input_value = false;
            }
            break;
        case InputKind::Right:
            if (x + 1 < image_x_dim) {
                int64_t right_index = offset + 1;
                waitUntilAvailable(neighbour_shared_values, neighbour_program_counter, neighbour_update_pc, right_index, image_size);
                input_value = neighbour_shared_values[right_index];
            } else {
                input_value = false;
            }
            break;
        case InputKind::Left:
            if (x - 1 >= 0) {
                int64_t left_index = offset - 1;
                waitUntilAvailable(neighbour_shared_values, neighbour_program_counter, neighbour_update_pc, left_index, image_size);
                input_value = neighbour_shared_values[left_index];
            } else {
                input_value = false;
            }
            break;
        default:
            break;
    }
    return (inputc.negated) ? !input_value : input_value;
}

__global__ void processingElemKernel(
    Instruction* instructions,
    size_t num_instructions,
    uint8_t* image,
    bool* neighbour_shared_values,
    size_t* neighbour_program_counter,
    bool* external_values,
    size_t image_size,
    size_t image_x_dim,
    size_t image_y_dim,
    size_t num_outputs
) {
    size_t x = threadIdx.x + blockIdx.x * blockDim.x;
    size_t y = threadIdx.y + blockIdx.y * blockDim.y;
    size_t offset = x + y * blockDim.x * gridDim.x;

    if (offset < image_size) {
        // image_x, image_y in image space
        // x, y in thread/block space
        size_t image_x = offset % image_x_dim;
        size_t image_y = offset / image_x_dim;
        const size_t MEMORY_SIZE_IN_BITS = 24;
        bool memory[MEMORY_SIZE_IN_BITS];
        for (size_t i = 0; i < MEMORY_SIZE_IN_BITS; i++) memory[i] = false;
        bool carry_register = false;
        size_t pd_bit = 0;
        size_t output_number = 0;

        // TODO DONT FORGET TO UPDATE
        size_t neighbour_update_pc = 0;

        for (size_t i = 0; i < num_instructions; i++) {
            const Instruction instruction = instructions[i];
            size_t pc = i + 1;
            bool carryval = false;
            switch (instruction.carry) {
                case Carry::CR: carryval = carry_register; break;
                case Carry::One: carryval = true; break;
                case Carry::Zero: carryval = false; break;
            }
            bool input_one = getInstructionInputValue(
                instruction.input1,
                memory,
                image,
                pd_bit,
                image_x,
                image_y,
                image_x_dim,
                image_y_dim,
                image_size,
                offset,
                neighbour_program_counter,
                neighbour_shared_values,
                neighbour_update_pc
            );
            bool input_two = getInstructionInputValue(
                instruction.input2,
                memory,
                image,
                pd_bit,
                image_x,
                image_y,
                image_x_dim,
                image_y_dim,
                image_size,
                offset,
                neighbour_program_counter,
                neighbour_shared_values,
                neighbour_update_pc
            );

            // printf("offset: %lu, instruction: %lu, input_one: %d, carryval: %d, input_two: %d\n", offset, i, input_one, carryval, input_two);

            const bool sum = (input_one != input_two) != carryval;
            const bool carry = (carryval && (input_one != input_two)) || (input_one && input_two);

            // TODO assuming can only be two values
            bool resultvalue = (instruction.resultType.value == 's') ? sum : carry;

            // Interesting choice...
            if (instruction.carry == Carry::CR) {
                carry_register = carry;
            }

            switch (instruction.result.resultKind) {
                case ResultKind::Address:
                    memory[instruction.result.address] = resultvalue;
                    break;
                case ResultKind::Neighbour:
                    neighbour_update_pc = pc;
                    neighbour_shared_values[offset] = resultvalue;
                    neighbour_program_counter[offset] = pc;
                    break;
                case ResultKind::External:
                    external_values[num_outputs * offset + output_number] = resultvalue;
                    output_number++;
                    break;
            }
        }
    }
};
