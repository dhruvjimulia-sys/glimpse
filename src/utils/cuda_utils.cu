#include "hip/hip_runtime.h"
#include <iostream>
#include "cuda_utils.h"

void queryGPUProperties() {
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    HANDLE_ERROR(err);

    if (deviceCount == 0) {
        std::cout << "No CUDA-capable devices found." << std::endl;
        return;
    }

    std::cout << "Number of CUDA-capable devices: " << deviceCount << std::endl;

    for (int device = 0; device < deviceCount; ++device) {
        hipDeviceProp_t deviceProp;
        HANDLE_ERROR(hipGetDeviceProperties(&deviceProp, device));

        std::cout << "\nDevice " << device << ": " << deviceProp.name << std::endl;
        std::cout << "  Compute capability: " << deviceProp.major << "." << deviceProp.minor << std::endl;
        std::cout << "  Total global memory: " << deviceProp.totalGlobalMem / (1024 * 1024) << " MB" << std::endl;
        std::cout << "  Shared memory per block: " << deviceProp.sharedMemPerBlock / 1024 << " KB" << std::endl;
        std::cout << "  Registers per block: " << deviceProp.regsPerBlock << std::endl;
        std::cout << "  Warp size: " << deviceProp.warpSize << std::endl;
        std::cout << "  Total constant memory: " << deviceProp.totalConstMem / 1024 << " KB" << std::endl;
        std::cout << "  Multiprocessors: " << deviceProp.multiProcessorCount << std::endl;
        std::cout << "  CUDA Cores/MP: " << (deviceProp.major == 8 ? 128 : 64) << " (Architecture-dependent)" << std::endl;
        std::cout << "  Total CUDA Cores: " << deviceProp.multiProcessorCount * (deviceProp.major == 8 ? 128 : 64) << std::endl;
        std::cout << "  Warp size: " << deviceProp.warpSize << std::endl;
        std::cout << "  Max threads per block: " << deviceProp.maxThreadsPerBlock << std::endl;
        std::cout << "  Max thread dimensions: ["
                  << deviceProp.maxThreadsDim[0] << ", "
                  << deviceProp.maxThreadsDim[1] << ", "
                  << deviceProp.maxThreadsDim[2] << "]" << std::endl;
        std::cout << "  Max grid size: ["
                  << deviceProp.maxGridSize[0] << ", "
                  << deviceProp.maxGridSize[1] << ", "
                  << deviceProp.maxGridSize[2] << "]" << std::endl;
        std::cout << "  Memory clock rate: " << deviceProp.memoryClockRate / 1000 << " MHz" << std::endl;
        std::cout << "  Memory bus width: " << deviceProp.memoryBusWidth << " bits" << std::endl;
        std::cout << "  Peak memory bandwidth: "
                  << 2.0 * deviceProp.memoryClockRate * (deviceProp.memoryBusWidth / 8) / 1.0e6 << " GB/s" << std::endl;
    }
}
