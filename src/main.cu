#include <iostream>
#include "main.h"
#include "isa.h"
#include "pe.h"
#include "utils/cuda_utils.h"
#include "utils/file_utils.h"
#include "utils/program_utils.h"

#define STB_IMAGE_IMPLEMENTATION
#include "stb/stb_image.h"
#define STB_IMAGE_RESIZE_IMPLEMENTATION
#include "stb/stb_image_resize2.h"


uint8_t* transform_image(const char* filename, int new_dimension, int new_bits) {
    int width, height, channels;
    uint8_t* img_data = stbi_load(filename, &width, &height, &channels, 0);
    if (!img_data) {
        return nullptr;
    }

    if (new_bits < 1 || new_bits > 8) {
        stbi_image_free(img_data);
        return nullptr;
    }

    // TODO Make resize automatic
    uint8_t* resized_data = img_data;
    
    // (uint8_t*)malloc(new_dimension * new_dimension * channels);
    // if (!resized_data) {
    //     stbi_image_free(img_data);
    //     return nullptr;
    // }

    // // Resize the image
    // stbir_resize_uint8(img_data, width, height, 0,
    //                    resized_data, new_dimension, new_dimension, 0,
    //                    channels);
    // stbi_image_free(img_data); // Free original image data

    // Convert to grayscale (1 channel)
    uint8_t* gray_data = (uint8_t*)malloc(new_dimension * new_dimension);
    if (!gray_data) {
        free(resized_data);
        return nullptr;
    }

    for (int i = 0; i < new_dimension * new_dimension; ++i) {
        int src_idx = i * channels;
        if (channels >= 3) {
            // Use luminance formula: 0.299*R + 0.587*G + 0.114*B (integer approximation)
            uint8_t r = resized_data[src_idx];
            uint8_t g = resized_data[src_idx + 1];
            uint8_t b = resized_data[src_idx + 2];
            gray_data[i] = static_cast<uint8_t>((r * 299 + g * 587 + b * 114 + 500) / 1000);
        } else {
            gray_data[i] = resized_data[src_idx];
        }
    }
    free(resized_data); 

    // Quantized to required bit depth
    const int max_level = (1 << new_bits) - 1;
    if (max_level > 0) {
        for (int i = 0; i < new_dimension * new_dimension; ++i) {
            gray_data[i] = (gray_data[i] >> (8 - new_bits)) & max_level;
        }
    }

    return gray_data;
}

bool *processImage(Program program, uint8_t* pixels, size_t image_x_dim, size_t image_y_dim) {
    size_t program_num_outputs = numOutputs(program);

    // Maximum of value below is 32
    size_t num_threads_per_block_per_dim = 16;
    
    // TODO make this CUDA memory constant as optimization
    Instruction* dev_instructions;
    size_t instructions_mem_size = sizeof(Instruction) * program.instructionCount;
    HANDLE_ERROR(hipMalloc((void **) &dev_instructions, instructions_mem_size));
    HANDLE_ERROR(hipMemcpy(dev_instructions, program.instructions, instructions_mem_size, hipMemcpyHostToDevice));

    // read grayscale pixels from image and memcpy to cuda memory
    // TODO make this CUDA memory constant as optimization

    size_t image_size = image_x_dim * image_y_dim;
    
    size_t image_mem_size = sizeof(uint8_t) * image_size;

    uint8_t* dev_image;

    HANDLE_ERROR(hipMalloc((void **) &dev_image, image_mem_size));
    HANDLE_ERROR(hipMemcpy(dev_image, pixels, image_mem_size, hipMemcpyHostToDevice));

    // neighbour
    bool* dev_neighbour_shared_values;
    size_t neighbour_shared_mem_size = sizeof(bool) * image_size;
    HANDLE_ERROR(hipMalloc((void **) &dev_neighbour_shared_values, neighbour_shared_mem_size));
    HANDLE_ERROR(hipMemset(dev_neighbour_shared_values, 0, neighbour_shared_mem_size));

    // program counter when neighbour written
    size_t* dev_neighbour_program_counter;
    size_t neighbour_program_counter_mem_size = sizeof(size_t) * image_size;
    HANDLE_ERROR(hipMalloc((void **) &dev_neighbour_program_counter, neighbour_program_counter_mem_size));
    HANDLE_ERROR(hipMemset(dev_neighbour_program_counter, 0, neighbour_program_counter_mem_size));

    // external values
    bool* dev_external_values;
    size_t external_values_mem_size = sizeof(bool) * image_size * program_num_outputs;
    HANDLE_ERROR(hipMalloc((void **) &dev_external_values, external_values_mem_size));
    HANDLE_ERROR(hipMemset(dev_external_values, 0, external_values_mem_size));

    dim3 blocks(
        (image_x_dim + num_threads_per_block_per_dim - 1) / num_threads_per_block_per_dim,
        (image_y_dim + num_threads_per_block_per_dim - 1) / num_threads_per_block_per_dim
    );
    dim3 threads(num_threads_per_block_per_dim, num_threads_per_block_per_dim);
    processingElemKernel<<<blocks, threads>>>(
        dev_instructions,
        program.instructionCount,
        dev_image,
        dev_neighbour_shared_values,
        dev_neighbour_program_counter,
        dev_external_values,
        image_size,
        image_x_dim,
        image_y_dim,
        program_num_outputs
    );

    HANDLE_ERROR(hipPeekAtLastError());

    HANDLE_ERROR(hipDeviceSynchronize());

    bool* external_values = (bool *) malloc(external_values_mem_size);
    HANDLE_ERROR(hipMemcpy(external_values, dev_external_values, external_values_mem_size, hipMemcpyDeviceToHost));

    HANDLE_ERROR(hipFree(dev_instructions));
    HANDLE_ERROR(hipFree(dev_image));
    HANDLE_ERROR(hipFree(dev_neighbour_shared_values));
    HANDLE_ERROR(hipFree(dev_neighbour_program_counter));
    HANDLE_ERROR(hipFree(dev_external_values));

    return external_values;
}


int main() {
    queryGPUProperties();

    std::string programFilename = "programs/edge_detection_one_bit.vis";
    const char *imageFilename = "images/windmill_resized.jpg";

    size_t dimension = 128;
    size_t num_bits = 1;

    uint8_t* image = transform_image(imageFilename, dimension, num_bits);
    
    // print image
    // for (size_t i = 0; i < dimension; i++) {
    //     for (size_t j = 0; j < dimension; j++) {
    //         printf("%d ", image[i * dimension + j]);
    //     }
    //     printf("\n");
    // }

    std::string programText;
    readFile(programFilename, programText);

    Parser parser(programText);
    Program program = parser.parse();
    program.print();

    size_t program_num_outputs = numOutputs(program);

    std::vector<std::vector<std::vector<bool>>> expected_image(dimension, std::vector<std::vector<bool>>(dimension, std::vector<bool>(program_num_outputs, 0)));
    for (int i = 0; i < dimension; i++) {
        for (int j = 0; j < dimension; j++) {
            size_t val = image[i * dimension + j];
            expected_image[i][j][0] =
            (((i - 1 < 0) ? 0 : image[(i - 1) * dimension + j]) != val)
            || (((i + 1 >= dimension) ? 0 : image[(i + 1) * dimension + j]) != val)
            || (((j - 1 < 0) ? 0 : image[i * dimension + (j - 1)]) != val)
            || (((j + 1 >= dimension) ? 0 : image[i * dimension + j + 1]) != val); 
        }
    }

    bool* processed_image = processImage(program, image, dimension, dimension);

    bool test_passed = true;
    for (size_t y = 0; y < dimension; y++) {
        for (size_t x = 0; x < dimension; x++) {
            size_t offset = x + y * dimension;
            for (int64_t i = program_num_outputs - 1; i >= 0; i--) {
                bool actual_value = processed_image[program_num_outputs * offset + i];
                if (actual_value != expected_image[y][x][i]) {
                    test_passed = false;
                }
            }
        }
    }

    if (test_passed) {
        std::cout << "Test passed" << std::endl;
    } else {
        std::cout << "Test failed" << std::endl;
    }

    free(image);
    free(program.instructions);
    return EXIT_SUCCESS;
}