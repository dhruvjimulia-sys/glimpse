#include <iostream>
#include "main.h"
#include "isa.h"
#include "pe.h"
#include "utils/cuda_utils.h"
#include "utils/file_utils.h"
#include "utils/program_utils.h"

int main() {
    queryGPUProperties();

    std::string programFilename = "programs/edge_detection_one_bit.vis";
    std::string imageFilename = "";

    // Maximum of value below is 32
    size_t num_threads_per_block_per_dim = 16;

    // read instructions from file, parse and memcpy to cuda memory
    // TODO make this CUDA memory constant as optimization
    std::string programText;
    readFile(programFilename, programText);

    Parser parser(programText);
    Program program = parser.parse();
    program.print();

    size_t program_num_outputs = numOutputs(program);

    Instruction* dev_instructions;
    size_t instructions_mem_size = sizeof(Instruction) * program.instructionCount;
    HANDLE_ERROR(hipMalloc((void **) &dev_instructions, instructions_mem_size));
    HANDLE_ERROR(hipMemcpy(dev_instructions, program.instructions, instructions_mem_size, hipMemcpyHostToDevice));

    // read grayscale pixels from image and memcpy to cuda memory
    // TODO make this CUDA memory constant as optimization
    size_t image_x_dim = 20;
    size_t image_y_dim = 20;
    size_t image_size = image_x_dim * image_y_dim;
    
    size_t image_mem_size = sizeof(uint8_t) * image_size;
    // uint8_t* pixels = (uint8_t*) malloc(image_mem_size);
    
    // // TODO For now, make all pixels one
    // for (size_t i = 0; i < image_size; i++) {
    //     pixels[i] = 1;
    // }

    uint8_t pixels[20][20] = {
        {0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0},
        {0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0},
        {0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0},
        {0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0},
        {0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0},
        {0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0},
        {0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0},
        {0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0},
        {1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0},
        {1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0},
        {1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0},
        {1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0},
        {0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0},
        {0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0},
        {0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0},
        {0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0},
        {0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0},
        {0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0},
        {0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0},
        {0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0},
    };

    // uint8_t pixels[20][20] = {
    //     {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
    //     {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
    //     {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
    //     {0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0},
    //     {0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0},
    //     {0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0},
    //     {0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0},
    //     {0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0},
    //     {0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0},
    //     {0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
    //     {0, 0, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
    //     {0, 0, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
    //     {0, 0, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
    //     {0, 0, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
    //     {0, 0, 0, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
    //     {0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
    //     {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
    //     {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
    //     {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
    //     {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
    // };

    uint8_t* dev_image;

    HANDLE_ERROR(hipMalloc((void **) &dev_image, image_mem_size));
    HANDLE_ERROR(hipMemcpy(dev_image, pixels, image_mem_size, hipMemcpyHostToDevice));

    // neighbour
    bool* dev_neighbour_shared_values;
    size_t neighbour_shared_mem_size = sizeof(bool) * image_size;
    HANDLE_ERROR(hipMalloc((void **) &dev_neighbour_shared_values, neighbour_shared_mem_size));
    HANDLE_ERROR(hipMemset(dev_neighbour_shared_values, 0, neighbour_shared_mem_size));

    // program counter when neighbour written
    size_t* dev_neighbour_program_counter;
    size_t neighbour_program_counter_mem_size = sizeof(size_t) * image_size;
    HANDLE_ERROR(hipMalloc((void **) &dev_neighbour_program_counter, neighbour_program_counter_mem_size));
    HANDLE_ERROR(hipMemset(dev_neighbour_program_counter, 0, neighbour_program_counter_mem_size));

    // external values
    bool* dev_external_values;
    size_t external_values_mem_size = sizeof(bool) * image_size * program_num_outputs;
    HANDLE_ERROR(hipMalloc((void **) &dev_external_values, external_values_mem_size));
    HANDLE_ERROR(hipMemset(dev_external_values, 0, external_values_mem_size));

    // TODO generalize to arbitrary length messages
    dim3 blocks(
        (image_x_dim + num_threads_per_block_per_dim - 1) / num_threads_per_block_per_dim,
        (image_y_dim + num_threads_per_block_per_dim - 1) / num_threads_per_block_per_dim
    );
    dim3 threads(num_threads_per_block_per_dim, num_threads_per_block_per_dim);
    processingElemKernel<<<blocks, threads>>>(
        dev_instructions,
        program.instructionCount,
        dev_image,
        dev_neighbour_shared_values,
        dev_neighbour_program_counter,
        dev_external_values,
        image_size,
        image_x_dim,
        image_y_dim,
        program_num_outputs
    );

    HANDLE_ERROR(hipDeviceSynchronize());

    bool* external_values = (bool *) malloc(external_values_mem_size);
    HANDLE_ERROR(hipMemcpy(external_values, dev_external_values, external_values_mem_size, hipMemcpyDeviceToHost));

    for (size_t y = 0; y < image_y_dim; y++) {
        for (size_t x = 0; x < image_x_dim; x++) {
            size_t offset = x + y * image_x_dim;
            for (int64_t i = program_num_outputs - 1; i >= 0; i--) {
                printf("%d", external_values[program_num_outputs * offset + i]);
            }
            printf(" ");
        }
        printf("\n");
    }

    HANDLE_ERROR(hipFree(dev_instructions));
    HANDLE_ERROR(hipFree(dev_image));
    HANDLE_ERROR(hipFree(dev_neighbour_shared_values));
    HANDLE_ERROR(hipFree(dev_neighbour_program_counter));
    HANDLE_ERROR(hipFree(dev_external_values));

    // free(pixels);
    return EXIT_SUCCESS;
}