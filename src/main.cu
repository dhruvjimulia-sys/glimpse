#include "hip/hip_runtime.h"
#include <iostream>
#include "main.h"
#include "isa.h"
#include "utils/cuda_utils.h"
#include "utils/file_utils.h"
#include "utils/program_utils.h"

__device__ bool getBitAt(uint8_t pixel_value, size_t bit_num) {
    if (bit_num >= 8) {
        printf("PD called more times than number of bits in image");
        return 0;
    } else {
        return pixel_value & (1 << bit_num); 
    }
}

__device__ void waitUntilAvailable(
    size_t* neighbour_program_counter,
    size_t neighbour_pc,
    size_t index,
    size_t image_size
) {
    while (index >= 0 && index < image_size && neighbour_program_counter[index] < neighbour_pc);
}

__device__ bool getInstructionInputValue(
    InputC inputc,
    bool *memory,
    uint8_t* image,
    size_t pd_bit,
    size_t image_x_dim,
    size_t image_size,
    size_t offset,
    size_t* neighbour_program_counter,
    bool* neighbour_shared_values,
    size_t neighbour_update_pc
) {
    bool input_value = false;
    switch (inputc.input.inputKind) {
        case InputKind::Address: input_value = memory[inputc.input.address]; break;
        case InputKind::ZeroValue: input_value = false; break;
        case InputKind::PD: input_value = getBitAt(image[offset], pd_bit); pd_bit++; break;
        case InputKind::Up:
            size_t up_index = offset - image_x_dim;
            waitUntilAvailable(neighbour_program_counter, neighbour_update_pc, up_index, image_size);
            input_value = (up_index >= 0) ? neighbour_shared_values[up_index] : 0;
            break;
        case InputKind::Down:
            size_t down_index = offset + image_x_dim;
            waitUntilAvailable(neighbour_program_counter, neighbour_update_pc, down_index, image_size);
            input_value = (down_index < image_size) ? neighbour_shared_values[down_index] : 0;
            break;
        case InputKind::Right:
            size_t right_index = offset + 1;
            waitUntilAvailable(neighbour_program_counter, neighbour_update_pc, right_index, image_size);
            input_value = (right_index < image_size) ? neighbour_shared_values[right_index] : 0;
            break;
        case InputKind::Left:
            size_t left_index = offset - 1;
            waitUntilAvailable(neighbour_program_counter, neighbour_update_pc, left_index, image_size);
            input_value = (left_index >= 0) ? neighbour_shared_values[left_index] : 0;
            break;
    }
    return (inputc.negated) ? ~input_value : input_value;
}

__global__ void processingElemKernel(
    Instruction* instructions,
    size_t num_instructions,
    uint8_t* image,
    bool* neighbour_shared_values,
    size_t* neighbour_program_counter,
    bool* external_values,
    size_t image_size,
    size_t image_x_dim,
    size_t image_y_dim,
    size_t num_outputs
) {
    size_t x = threadIdx.x + blockIdx.x * blockDim.x;
    size_t y = threadIdx.y + blockIdx.y * blockDim.y;
    size_t offset = x + y * blockDim.x * gridDim.x;

    if (offset < image_size) {
        const size_t MEMORY_SIZE_IN_BITS = 24;
        bool memory[MEMORY_SIZE_IN_BITS];
        for (size_t i = 0; i < MEMORY_SIZE_IN_BITS; i++) memory[i] = false;
        bool carry_register = false;
        size_t pd_bit = 0;
        size_t output_number = 0;

        // TODO DONT FORGET TO UPDATE
        size_t neighbour_update_pc = 0;

        for (size_t i = 0; i < num_instructions; i++) {
            const Instruction instruction = instructions[i];
            size_t pc = i + 1;
            bool carryval = false;
            switch (instruction.carry) {
                case Carry::CR: carryval = carry_register; break;
                case Carry::One: carryval = true; break;
                case Carry::Zero: carryval = false; break;
            }
            bool input_one = getInstructionInputValue(
                instruction.input1,
                memory,
                image,
                pd_bit,
                image_x_dim,
                image_size,
                offset,
                neighbour_program_counter,
                neighbour_shared_values,
                neighbour_update_pc
            );
            bool input_two = getInstructionInputValue(
                instruction.input2,
                memory,
                image,
                pd_bit,
                image_x_dim,
                image_size,
                offset,
                neighbour_program_counter,
                neighbour_shared_values,
                neighbour_update_pc
            );

            const bool sum = (input_one != input_two) != carryval;
            const bool carry = (carryval && (input_one != input_two)) || (input_one && input_two);

            // TODO assuming can only be two values
            bool resultvalue = (instruction.resultType.value == 's') ? sum : carry;

            // TODO Only update carry register when carry set to CR?
            carry_register = carry;

            switch (instruction.result.resultKind) {
                case ResultKind::Address:
                    memory[instruction.result.address] = resultvalue;
                    break;
                case ResultKind::Neighbour:
                    neighbour_update_pc = pc;
                    neighbour_shared_values[offset] = resultvalue;
                    neighbour_program_counter[offset] = pc;
                    break;
                case ResultKind::External:
                    external_values[num_outputs * offset + output_number] = resultvalue;
                    output_number++;
                    break;
            }
        }
    }
};

int main() {
    queryGPUProperties();

    std::string programFilename = "programs/prewitt.vis";
    std::string imageFilename = "";

    // Maximum of value below is 32
    size_t num_threads_per_block_per_dim = 16;

    size_t image_x_dim = 1;
    size_t image_y_dim = 1;
    size_t image_size = image_x_dim * image_y_dim;

    // read instructions from file, parse and memcpy to cuda memory
    // TODO make this CUDA memory constant as optimization
    std::string programText;
    readFile(programFilename, programText);

    Parser parser(programText);
    Program program = parser.parse();
    program.print();

    size_t program_num_outputs = numOutputs(program);

    Instruction* dev_instructions;
    size_t instructions_mem_size = sizeof(Instruction) * program.instructionCount;
    HANDLE_ERROR(hipMalloc((void **) &dev_instructions, instructions_mem_size));
    HANDLE_ERROR(hipMemcpy(dev_instructions, program.instructions, instructions_mem_size, hipMemcpyHostToDevice));

    // read grayscale pixels from image and memcpy to cuda memory
    // TODO make this CUDA memory constant as optimization
    size_t image_mem_size = sizeof(uint8_t) * image_size;
    uint8_t* pixels = (uint8_t*) malloc(image_mem_size);
    
    // TODO For now, make all pixels one
    for (size_t i = 0; i < image_size; i++) {
        pixels[i] = 1;
    }
    uint8_t* dev_image;
    HANDLE_ERROR(hipMalloc((void **) &dev_image, image_mem_size));
    HANDLE_ERROR(hipMemcpy(dev_image, pixels, image_mem_size, hipMemcpyHostToDevice));

    // neighbour
    bool* dev_neighbour_shared_values;
    size_t neighbour_shared_mem_size = sizeof(bool) * image_size;
    HANDLE_ERROR(hipMalloc((void **) &dev_neighbour_shared_values, neighbour_shared_mem_size));
    HANDLE_ERROR(hipMemset(dev_neighbour_shared_values, 0, neighbour_shared_mem_size));

    // program counter when neighbour written
    size_t* dev_neighbour_program_counter;
    size_t neighbour_program_counter_mem_size = sizeof(size_t) * image_size;
    HANDLE_ERROR(hipMalloc((void **) &dev_neighbour_program_counter, neighbour_program_counter_mem_size));
    HANDLE_ERROR(hipMemset(dev_neighbour_program_counter, 0, neighbour_program_counter_mem_size));

    // external values
    bool* dev_external_values;
    size_t external_values_mem_size = sizeof(bool) * image_size * program_num_outputs;
    HANDLE_ERROR(hipMalloc((void **) &dev_external_values, external_values_mem_size));
    HANDLE_ERROR(hipMemset(dev_external_values, 0, external_values_mem_size));

    // TODO generalize to arbitrary length messages
    dim3 blocks(
        (image_x_dim + num_threads_per_block_per_dim - 1) / num_threads_per_block_per_dim,
        (image_y_dim + num_threads_per_block_per_dim - 1) / num_threads_per_block_per_dim
    );
    dim3 threads(num_threads_per_block_per_dim, num_threads_per_block_per_dim);
    processingElemKernel<<<blocks, threads>>>(
        dev_instructions,
        program.instructionCount,
        dev_image,
        dev_neighbour_shared_values,
        dev_neighbour_program_counter,
        dev_external_values,
        image_size,
        image_x_dim,
        image_y_dim,
        program_num_outputs
    );

    hipDeviceSynchronize();

    HANDLE_ERROR(hipFree(dev_instructions));
    HANDLE_ERROR(hipFree(dev_image));
    HANDLE_ERROR(hipFree(dev_neighbour_shared_values));
    HANDLE_ERROR(hipFree(dev_neighbour_program_counter));
    HANDLE_ERROR(hipFree(dev_external_values));

    free(pixels);
    return EXIT_SUCCESS;
}