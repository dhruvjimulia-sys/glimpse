#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <cuda/atomic>
#include "main.h"
#include "isa.h"
#include "pe.h"
#include "utils/cuda_utils.h"
#include "utils/file_utils.h"
#include "utils/program_utils.h"
#include "powerandarea.h"
#include <sys/stat.h>
#include <sys/types.h>
#include <errno.h>

#define STB_IMAGE_IMPLEMENTATION
#include "stb/stb_image.h"
#define STB_IMAGE_RESIZE_IMPLEMENTATION
#include "stb/stb_image_resize2.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb/stb_image_write.h"

// Note: can move to utils
std::string replaceAll(const std::string& input,
    const std::string& from,
    const std::string& to) {
    if (from.empty()) return input;  // Avoid infinite loop

    std::string result = input;
    size_t startPos = 0;

    while ((startPos = result.find(from, startPos)) != std::string::npos) {
        result.replace(startPos, from.length(), to);
        startPos += to.length();  // Move past the replacement
    }

    return result;
}

uint8_t quantizeTo8Bit(uint16_t x, uint8_t n) {
    if (n >= 16 || n == 0) {
        throw std::invalid_argument("n must be between 1 and 15");
    }

    uint32_t max_val = (1u << n) - 1;    // Max value for n bits
    x = std::min(x, static_cast<uint16_t>(max_val));  // Clip to [0, 2^n - 1]
    
    // Round to nearest integer by adding half of max_val
    uint32_t result = (static_cast<uint32_t>(x) * 255 + (max_val / 2)) / max_val;
    return static_cast<uint8_t>(result);
}

uint8_t* transform_image(const char* filename, int new_dimension, int new_bits) {
    int width, height, channels;
    // Check if the file exists
    FILE *file = fopen(filename, "r");
    if (!file) {
        std::cerr << "Error: Could not open file " << filename << std::endl;
        exit(EXIT_SUCCESS);
    } else {
        fclose(file);
    }

    uint8_t* img_data = stbi_load(filename, &width, &height, &channels, 0);
    if (!img_data) {
        return nullptr;
    }

    if (new_bits < 1 || new_bits > 8) {
        stbi_image_free(img_data);
        return nullptr;
    }
    
    uint8_t* resized_data = (uint8_t*)malloc(new_dimension * new_dimension * channels);
    if (!resized_data) {
        stbi_image_free(img_data);
        return nullptr;
    }

    // Resize the image
    stbir_resize_uint8_linear(img_data, width, height, 0,
                       resized_data, new_dimension, new_dimension, 0,
                       (stbir_pixel_layout) channels);
    stbi_image_free(img_data); // Free original image data

    // Convert to grayscale (1 channel)
    uint8_t* gray_data = (uint8_t*)malloc(new_dimension * new_dimension);
    if (!gray_data) {
        free(resized_data);
        return nullptr;
    }

    for (int i = 0; i < new_dimension * new_dimension; ++i) {
        int src_idx = i * channels;
        if (channels >= 3) {
            // Use luminance formula: 0.299*R + 0.587*G + 0.114*B (integer approximation)
            uint8_t r = resized_data[src_idx];
            uint8_t g = resized_data[src_idx + 1];
            uint8_t b = resized_data[src_idx + 2];
            gray_data[i] = static_cast<uint8_t>((r * 299 + g * 587 + b * 114 + 500) / 1000);
        } else {
            gray_data[i] = resized_data[src_idx];
        }
    }
    free(resized_data); 

    // Quantized to required bit depth
    const int max_level = (1 << new_bits) - 1;
    if (max_level > 0) {
        for (int i = 0; i < new_dimension * new_dimension; ++i) {
            gray_data[i] = (gray_data[i] >> (8 - new_bits)) & max_level;
        }
    }

    // Create "outputimages" directory if it doesn't exist
    if (mkdir("outputimages", 0777) == -1) {
        if (errno != EEXIST) {
            std::cerr << "Error creating directory: " << strerror(errno) << std::endl;
        }
    }

    // Save the grayscale image
    uint8_t* image_output_data = (uint8_t*) malloc(new_dimension * new_dimension * sizeof(uint8_t));
    for (int i = 0; i < new_dimension * new_dimension; ++i) {
        image_output_data[i] = quantizeTo8Bit(gray_data[i], new_bits);
    }
    std::string output_filename = "outputimages/visionchipimg_" + std::to_string(new_bits);
    stbi_write_png((output_filename + ".png").c_str(), new_dimension, new_dimension, 1, image_output_data, new_dimension);
    free(image_output_data);

    return gray_data;
}

std::pair<bool *, float> process_image_gpu(Program program, uint8_t* pixels, size_t image_x_dim, size_t image_y_dim, size_t num_iterations) {
    size_t program_num_outputs = numOutputs(program);
    size_t program_num_shared_neighbours = numSharedNeighbours(program);
    
    // Non-constant memory version
    Instruction* dev_instructions;
    size_t instructions_mem_size = sizeof(Instruction) * program.instructionCount * program.vliwWidth;
    HANDLE_ERROR(hipMalloc((void **) &dev_instructions, instructions_mem_size));
    HANDLE_ERROR(hipMemcpy(dev_instructions, program.instructions, instructions_mem_size, hipMemcpyHostToDevice));

    // read grayscale pixels from image and memcpy to cuda memory
    size_t image_size = image_x_dim * image_y_dim;
    size_t image_mem_size = sizeof(uint8_t) * image_size;
    uint8_t* dev_image;
    HANDLE_ERROR(hipMalloc((void **) &dev_image, image_mem_size));
    HANDLE_ERROR(hipMemcpy(dev_image, pixels, image_mem_size, hipMemcpyHostToDevice));

    // debugging output
    size_t* dev_debug_output = nullptr;
    size_t num_debug_outputs = 3;
    // size_t debug_output_mem_size = sizeof(size_t) * image_size * program.instructionCount * program.vliwWidth * num_debug_outputs;
    // HANDLE_ERROR(hipMalloc((void **) &dev_debug_output, debug_output_mem_size));
    // HANDLE_ERROR(hipMemset(dev_debug_output, 0, debug_output_mem_size));

    // neighbour
    bool* dev_neighbour_shared_values;
    size_t neighbour_shared_mem_size = sizeof(bool) * image_size * program_num_shared_neighbours;
    HANDLE_ERROR(hipMalloc((void **) &dev_neighbour_shared_values, neighbour_shared_mem_size));
    HANDLE_ERROR(hipMemset(dev_neighbour_shared_values, 0, neighbour_shared_mem_size));

    // external values
    bool* dev_external_values;
    size_t external_values_mem_size = sizeof(bool) * image_size * program_num_outputs * num_iterations;
    HANDLE_ERROR(hipMalloc((void **) &dev_external_values, external_values_mem_size));
    HANDLE_ERROR(hipMemset(dev_external_values, 0, external_values_mem_size));

    // local memory values
    bool* dev_local_memory_values;
    size_t local_memory_values_mem_size = sizeof(bool) * image_size * MEMORY_SIZE_IN_BITS;
    HANDLE_ERROR(hipMalloc((void **) &dev_local_memory_values, local_memory_values_mem_size));
    HANDLE_ERROR(hipMemset(dev_local_memory_values, 0, local_memory_values_mem_size));

    // carry register values
    bool* dev_carry_register_values;
    size_t carry_register_values_mem_size = sizeof(bool) * image_size * program.vliwWidth;
    HANDLE_ERROR(hipMalloc((void **) &dev_carry_register_values, carry_register_values_mem_size));
    HANDLE_ERROR(hipMemset(dev_carry_register_values, 0, carry_register_values_mem_size));

    // result values
    const size_t PIPELINE_WIDTH = 3;
    bool* dev_result_values;
    HANDLE_ERROR(hipMalloc((void **) &dev_result_values, image_size * PIPELINE_WIDTH * program.vliwWidth));

    hipEvent_t start, stop;
    float elapsedTime;
    
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));
        
    HANDLE_ERROR(hipEventRecord(start, 0));

    int numBlocksPerSm = 0;
    hipDeviceProp_t deviceProp;
    // ASSUME device = 0 is our device
    size_t device = 0;
    hipGetDeviceProperties(&deviceProp, device);
    size_t numThreads = NUM_THREADS_PER_BLOCK_PER_DIM * NUM_THREADS_PER_BLOCK_PER_DIM;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, processingElemKernel, numThreads, device);
    size_t MAX_BLOCK_SIZE = numBlocksPerSm * deviceProp.multiProcessorCount;

    dim3 blocks(
        std::min(((image_x_dim + NUM_THREADS_PER_BLOCK_PER_DIM - 1) / NUM_THREADS_PER_BLOCK_PER_DIM) * ((image_y_dim + NUM_THREADS_PER_BLOCK_PER_DIM - 1) / NUM_THREADS_PER_BLOCK_PER_DIM), MAX_BLOCK_SIZE),
        1
    );
    dim3 threads(NUM_THREADS_PER_BLOCK_PER_DIM, NUM_THREADS_PER_BLOCK_PER_DIM);

    void *kernelArgs[] = {
        (void *) &dev_instructions,
        (void *) &program.instructionCount,
        (void *) &dev_image,
        (void *) &dev_neighbour_shared_values,
        (void *) &dev_external_values,
        (void *) &image_size,
        (void *) &image_x_dim,
        (void *) &image_y_dim,
        (void *) &program_num_outputs,
        (void *) &program_num_shared_neighbours,
        (void *) &dev_debug_output,
        (void *) &num_debug_outputs,
        (void *) &program.vliwWidth,
        (void *) &program.isPipelining,
        (void *) &dev_local_memory_values,
        (void *) &dev_carry_register_values,
        (void *) &dev_result_values,
        (void *) &num_iterations
    };
    hipLaunchCooperativeKernel((void *) processingElemKernel, blocks, threads, kernelArgs);

    HANDLE_ERROR(hipPeekAtLastError());

    HANDLE_ERROR(hipDeviceSynchronize());

    HANDLE_ERROR(hipEventRecord(stop, 0));
    HANDLE_ERROR(hipEventSynchronize(stop));

    // Gets the elapsed time in milliseconds
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));

    bool* external_values = (bool *) malloc(external_values_mem_size);
    HANDLE_ERROR(hipMemcpy(external_values, dev_external_values, external_values_mem_size, hipMemcpyDeviceToHost));

    // debugging output
    // size_t* debug_output = (size_t *) malloc(debug_output_mem_size);
    // HANDLE_ERROR(hipMemcpy(debug_output, dev_debug_output, debug_output_mem_size, hipMemcpyDeviceToHost));
    // for (size_t i = 0; i < image_size; i++) {
    //     for (size_t j = 0; j < program.instructionCount * program.vliwWidth; j++) {
    //         size_t offset = (i * program.instructionCount * program.vliwWidth + j) * num_debug_outputs;
    //         std::cout << "Instruction " << j << " at " << i << ": ";
    //         for (size_t k = 0; k < num_debug_outputs; k++) {
    //             std::cout << debug_output[offset + k] << " ";
    //         }
    //         std::cout << std::endl;
    //     }
    // }

    HANDLE_ERROR(hipFree(dev_instructions));
    HANDLE_ERROR(hipFree(dev_image));
    HANDLE_ERROR(hipFree(dev_neighbour_shared_values));
    HANDLE_ERROR(hipFree(dev_external_values));
    HANDLE_ERROR(hipFree(dev_local_memory_values));
    HANDLE_ERROR(hipFree(dev_carry_register_values));
    HANDLE_ERROR(hipFree(dev_result_values));
    // HANDLE_ERROR(hipFree(dev_debug_output));

    return {external_values, elapsedTime};
}

bool get_instruction_input_value_cpu(
    InputC inputc,
    bool* memory,
    uint8_t* image,
    size_t pd_bit,
    bool* pd_increment,
    int64_t x,
    int64_t y,
    size_t image_x_dim,
    size_t image_y_dim,
    size_t image_size,
    size_t offset,
    bool* neighbour_shared_values,
    size_t num_shared_neighbours,
    size_t shared_neighbour_value
) {
    bool input_value = false;
    switch (inputc.input.inputKind) {
        case InputKind::Address: input_value = memory[inputc.input.address]; break;
        case InputKind::ZeroValue: input_value = false; break;
        case InputKind::PD:
            input_value = getBitAt(image[offset], pd_bit);
            *pd_increment = true;
            break;
        case InputKind::Up:
            if (y - 1 >= 0) {
                int64_t up_index = offset - image_x_dim;
                input_value = neighbour_shared_values[up_index * num_shared_neighbours + shared_neighbour_value - 1];
            } else {
                input_value = false;
            }
            break;
        case InputKind::Down:
            if (y + 1 < image_y_dim) {
                int64_t down_index = offset + image_x_dim;
                input_value = neighbour_shared_values[down_index * num_shared_neighbours + shared_neighbour_value - 1];
            } else {
                input_value = false;
            }
            break;
        case InputKind::Right:
            if (x + 1 < image_x_dim) {
                int64_t right_index = offset + 1;
                input_value = neighbour_shared_values[right_index * num_shared_neighbours + shared_neighbour_value - 1];
            } else {
                input_value = false;
            }
            break;
        case InputKind::Left:
            if (x - 1 >= 0) {
                int64_t left_index = offset - 1;
                input_value = neighbour_shared_values[left_index * num_shared_neighbours + shared_neighbour_value - 1];
            } else {
                input_value = false;
            }
            break;
        default:
            break;
    }
    return (inputc.negated) ? !input_value : input_value;
}

std::pair<bool *, float> process_image_cpu(Program program, uint8_t* pixels, size_t image_x_dim, size_t image_y_dim, size_t num_iterations) {
    size_t program_num_outputs = numOutputs(program);
    size_t program_num_shared_neighbours = numSharedNeighbours(program);
    size_t image_size = image_x_dim * image_y_dim;

    bool* neighbour_shared_values = (bool *) malloc(image_size * program_num_shared_neighbours);
    for (size_t i = 0; i < image_size * program_num_shared_neighbours; i++) {
        neighbour_shared_values[i] = false;
    }
    bool* local_memory_values = (bool *) malloc(image_size * MEMORY_SIZE_IN_BITS);
    for (size_t i = 0; i < image_size * MEMORY_SIZE_IN_BITS; i++) {
        local_memory_values[i] = false;
    }
    bool* carry_register = (bool *) malloc(image_size * program.vliwWidth);
    for (size_t i = 0; i < image_size * program.vliwWidth; i++) {
        carry_register[i] = false;
    }
    bool* external_values = (bool *) malloc(image_size * program_num_outputs * num_iterations);
    for (size_t i = 0; i < image_size * program_num_outputs * num_iterations; i++) {
        external_values[i] = false;
    }
    auto start_time = std::chrono::high_resolution_clock::now();

    for (size_t iter = 0; iter < num_iterations; iter++) {
        size_t output_number = 0;
        bool output_number_increment = false;
        size_t shared_neighbour_value = 0;
        bool shared_neighbour_increment = false;

        // Note: PIPELINE_WIDTH
        size_t PIPELINE_WIDTH = program.isPipelining ? 3 : 1;
        bool* result_values = (bool *) malloc(image_size * PIPELINE_WIDTH * program.vliwWidth);

        size_t pd_bit = 0;
        bool pd_increment = false;

        for (size_t i = 0; (i < program.instructionCount && !program.isPipelining) || (i < program.instructionCount + PIPELINE_WIDTH - 1 && program.isPipelining); i++) {
            for (size_t x = 0; x < image_x_dim; x++) {
                for (size_t y = 0; y < image_y_dim; y++) {
                    size_t offset = x + y * image_x_dim;
                    if (i < program.instructionCount) {
                        for (size_t j = 0; j < program.vliwWidth; j++) {
                            const Instruction instruction = program.instructions[i * program.vliwWidth + j];
                            if (instruction.isNop) {
                                continue;
                            }
                            bool carryval = false;
                            switch (instruction.carry) {
                                case Carry::CR: carryval = carry_register[offset * program.vliwWidth + j]; break;
                                case Carry::One: carryval = true; break;
                                case Carry::Zero: carryval = false; break;
                            }
                            bool input_one = get_instruction_input_value_cpu(
                                instruction.input1,
                                local_memory_values + offset * MEMORY_SIZE_IN_BITS,
                                pixels,
                                pd_bit,
                                &pd_increment,
                                x,
                                y,
                                image_x_dim,
                                image_y_dim,
                                image_size,
                                offset,
                                neighbour_shared_values,
                                program_num_shared_neighbours,
                                shared_neighbour_value
                            );
                            bool input_two = get_instruction_input_value_cpu(
                                instruction.input2,
                                local_memory_values + offset * MEMORY_SIZE_IN_BITS,
                                pixels,
                                pd_bit,
                                &pd_increment,
                                x,
                                y,
                                image_x_dim,
                                image_y_dim,
                                image_size,
                                offset,
                                neighbour_shared_values,
                                program_num_shared_neighbours,
                                shared_neighbour_value
                            );

                            const bool sum = (input_one != input_two) != carryval;
                            const bool carry = (carryval && (input_one != input_two)) || (input_one && input_two);
                            
                            result_values[(offset * PIPELINE_WIDTH + (i % PIPELINE_WIDTH)) * program.vliwWidth + j] = (instruction.resultType.value == 's') ? sum : carry;

                            // Interesting choice...
                            if (instruction.carry == Carry::CR) {
                                carry_register[offset * program.vliwWidth + j] = carry;
                            }
                        }
                    }

                    if (!program.isPipelining || (program.isPipelining && i >= PIPELINE_WIDTH - 1)) {
                        for (size_t j = 0; j < program.vliwWidth; j++) {
                            const Instruction instruction = 
                            program.isPipelining ?
                            program.instructions[(i - PIPELINE_WIDTH + 1) * program.vliwWidth + j] :
                            program.instructions[i * program.vliwWidth + j];
                            if (instruction.isNop) {
                                continue;
                            }
                            bool resultvalue = !program.isPipelining ?
                            result_values[(offset * PIPELINE_WIDTH + (i % PIPELINE_WIDTH)) * program.vliwWidth + j] :
                            result_values[(offset * PIPELINE_WIDTH + ((i - PIPELINE_WIDTH + 1) % PIPELINE_WIDTH)) * program.vliwWidth + j];
                            // result_values[offset * program.vliwWidth + j];
                            switch (instruction.result.resultKind) {
                                case ResultKind::Address:
                                    local_memory_values[offset * MEMORY_SIZE_IN_BITS + instruction.result.address] = resultvalue;
                                    break;
                                case ResultKind::Neighbour:
                                    neighbour_shared_values[offset * program_num_shared_neighbours + shared_neighbour_value] = resultvalue;
                                    shared_neighbour_increment = true;
                                    break;
                                case ResultKind::External:
                                    external_values[iter * program_num_outputs * image_size + program_num_outputs * offset + output_number] = resultvalue;
                                    output_number_increment = true;
                                    break;
                            }
                        }
                    }
                }
            }

            if (pd_increment) {
                pd_bit++;
            }
            pd_increment = false;

            if (shared_neighbour_increment) {
                shared_neighbour_value++;
            }
            shared_neighbour_increment = false;

            if (output_number_increment) {
                output_number++;
            }
            output_number_increment = false;
        }
    }

    auto stop_time = std::chrono::high_resolution_clock::now();
    size_t duration = std::chrono::duration_cast<std::chrono::microseconds>(stop_time - start_time).count();
    float durationInMilliseconds = duration / 1000.0f;

    free(neighbour_shared_values);
    free(local_memory_values);
    free(carry_register);

    // std::cout << "External values" << std::endl;
    // for (size_t i = 0; i < image_size * program_num_outputs; i++) {
    //     std::cout << "offset " << i << ": " << external_values[i] << std::endl;
    // }

    return {external_values, durationInMilliseconds};
}

void testProgram(std::string programFilename,
    size_t vliwWidth,
    bool isPipelining,
    const char *imageFilename,
    size_t dimension,
    size_t num_bits,
    size_t expected_program_num_outputs,
    size_t num_iterations,
    std::vector<std::vector<std::vector<std::vector<bool>>>> expected_image,
    std::vector<float>& real_time_timings,
    std::vector<float>& per_frame_timings,
    bool useGPU,
    bool test,
    bool twosComplementOutput
) {
    uint8_t* image = transform_image(imageFilename, dimension, num_bits);
    // Print image in binary form
    // std::cout << "Image (binary):" << std::endl;
    // for (size_t y = 0; y < dimension; y++) {
    //     for (size_t x = 0; x < dimension; x++) {
    //         for (size_t i = 0; i < num_bits; i++) {
    //             size_t val = (image[y * dimension + x] & (1 << i)) >> i;
    //             std::cout << val;
    //         }
    //         std::cout << " ";
    //     }
    //     std::cout << std::endl;
    // }

    // Print image
    // for (size_t y = 0; y < dimension; y++) {
    //     for (size_t x = 0; x < dimension; x++) {
    //         std::cout << (uint16_t) image[y * dimension + x] << " ";
    //     }
    //     std::cout << std::endl;
    // }

    // for (size_t y = 0; y < dimension; y++) {
    //     for (size_t x = 0; x < dimension; x++) {
    //         std::cout << "offset " << y * dimension + x << ": " << (int) image[y * dimension + x] << std::endl;
    //     }
    // }


    std::string programText;
    readFile(programFilename, programText);

    Parser parser(programText);
    Program program = parser.parse(vliwWidth, isPipelining);
    // program.print();

    size_t program_num_outputs = numOutputs(program);

    if (expected_program_num_outputs != program_num_outputs) {
        std::cerr << "Error: Expected program num outputs " << expected_program_num_outputs << " but got " << program_num_outputs << std::endl;
        exit(EXIT_FAILURE);
    }

    size_t image_size = dimension * dimension;

    bool *processed_image = nullptr;
    if (useGPU) {
        auto normal_start = std::chrono::high_resolution_clock::now();
        std::pair<bool *, float> process_image_result = process_image_gpu(program, image, dimension, dimension, num_iterations);
        auto normal_stop = std::chrono::high_resolution_clock::now();
        size_t real_time_duration = std::chrono::duration_cast<std::chrono::microseconds>(normal_stop - normal_start).count();
        processed_image = process_image_result.first;
        per_frame_timings.push_back(process_image_result.second / num_iterations);
        real_time_timings.push_back((real_time_duration / 1000.0f) / num_iterations);
    } else {
        auto normal_start = std::chrono::high_resolution_clock::now();
        std::pair<bool *, float> process_image_result = process_image_cpu(program, image, dimension, dimension, num_iterations);
        auto normal_stop = std::chrono::high_resolution_clock::now();
        size_t real_time_duration = std::chrono::duration_cast<std::chrono::microseconds>(normal_stop - normal_start).count();
        processed_image = process_image_result.first;
        per_frame_timings.push_back(process_image_result.second / num_iterations);
        real_time_timings.push_back((real_time_duration / 1000.0f) / num_iterations);
    }

    // Testing
    bool test_passed = true;
    if (test) {
        for (size_t iter = 0; iter < num_iterations; iter++) {
            for (size_t y = 0; y < dimension; y++) {
                for (size_t x = 0; x < dimension; x++) {
                    size_t offset = x + y * dimension;
                    for (int64_t i = program_num_outputs - 1; i >= 0; i--) {
                        bool actual_value = processed_image[iter * program_num_outputs * image_size + program_num_outputs * offset + i];
                        if (actual_value != expected_image[iter][y][x][i]) {
                            std::cout << "Mismatch at (" << x << ", " << y << ")[" << i << "] at iteration " << iter << ": " << actual_value << " != " << expected_image[iter][y][x][i] << std::endl;
                            test_passed = false;
                        }
                    }
                }
            }
        }
    }

    // Create "outputimages" directory if it doesn't exist
    if (mkdir("outputimages", 0777) == -1) {
        if (errno != EEXIST) {
            std::cerr << "Error creating directory: " << strerror(errno) << std::endl;
        }
    }

    std::string output_filename = "outputimages/" + replaceAll(replaceAll(programFilename, "/", "_"), ".", "_") + "_" + (useGPU ? "gpu" : "cpu");
    if (num_iterations > 1) {
        if (mkdir(output_filename.c_str(), 0777) == -1) {
            if (errno != EEXIST) {
                std::cerr << "Error creating directory: " << strerror(errno) << std::endl;
            }
        }
    }
    
    if (!twosComplementOutput) {
        uint8_t* data = (uint8_t*) malloc(dimension * dimension * sizeof(uint8_t));
        if (!data) {
            exit(EXIT_FAILURE); // Allocation failed
        }
        for (size_t iter = 0; iter < num_iterations; iter++) {
            // std::cout << "Iteration " << iter << ":" << std::endl;
            for (size_t y = 0; y < dimension; y++) {
                for (size_t x = 0; x < dimension; x++) {
                    size_t offset = x + y * dimension;
                    uint16_t val = 0;
                    // for (int i = program_num_outputs - 1; i >= 0; i--) {
                    //     std::cout << processed_image[iter * program_num_outputs * image_size + program_num_outputs * offset + i];
                    //     val |= processed_image[iter * program_num_outputs * image_size + program_num_outputs * offset + i] << i;
                    // }
                    const size_t MAX_BITS = 16;
                    for (int i = MAX_BITS - 1; i >= 0; i--) {
                        bool bit = (i < program_num_outputs) ? processed_image[iter * program_num_outputs * image_size + program_num_outputs * offset + i] : (
                            twosComplementOutput ?
                            processed_image[iter * program_num_outputs * image_size + program_num_outputs * offset + (program_num_outputs - 1)] :
                            0
                        );
                        if (i < program_num_outputs) {
                            // std::cout << bit;
                        }
                        val |= bit << i;
                    }
                    // printf("(%4d) ", (int16_t) val);
                    val = quantizeTo8Bit(val, program_num_outputs);
                    // std::cout << (int) val << " ";
                    data[y * dimension + x] = val;
                }
                // std::cout << std::endl;
            }
            if (num_iterations == 1) {
                stbi_write_png((output_filename + ".png").c_str(), dimension, dimension, 1, data, dimension);
            } else {
                std::string output_filename_iter = output_filename + "/iteration_" + std::to_string(iter) + ".png";
                stbi_write_png(output_filename_iter.c_str(), dimension, dimension, 1, data, dimension);
            }
        }
        free(data);
    }

    // Testing logging
    if (test_passed) {
        // Logging when tests pass
        std::cout << programFilename << " test passed with frame rate " << 1000.0f / per_frame_timings[per_frame_timings.size() - 1] << " fps" << std::endl;
    } else {
        std::cout << programFilename << " test failed" << std::endl;
    }

    // Print power and area
    /*
    double computeArea = getComputeArea(program.vliwWidth) * dimension * dimension;
    double memoryArea = getMemoryArea(program.vliwWidth, program.isPipelining) * dimension * dimension;
    double computeDynPower = getComputeDynamicPower(program) * dimension * dimension;
    double memoryDynPower = getMemoryDynamicPower(program) * dimension * dimension;
    double computeSubThreshLeakage = getComputeSubthresholdLeakage(program.vliwWidth) * dimension * dimension;
    double memorySubThreshLeakage = getMemorySubthresholdLeakage(program.vliwWidth, program.isPipelining) * dimension * dimension;
    double computeGateLeakage = getComputeGateLeakage(program.vliwWidth) * dimension * dimension;
    double memoryGateLeakage = getMemoryGateLeakage(program.vliwWidth, program.isPipelining) * dimension * dimension;

    // std::cout << "Compute Area: " << computeArea << " um^2" << std::endl;
    // std::cout << "Memory Area: " << memoryArea << " um^2" << std::endl;
    std::cout << std::fixed << "Area (um^2): " << computeArea + memoryArea << std::endl;
    // std::cout << "Compute Dynamic Power: " << computeDynPower << " W" << std::endl;
    // std::cout << "Memory Dynamic Power: " << memoryDynPower << " W" << std::endl;
    // std::cout << "Compute Subthreshold Leakage: " << computeSubThreshLeakage << " W" << std::endl;
    // std::cout << "Memory Subthreshold Leakage: " << memorySubThreshLeakage << " W" << std::endl;
    // std::cout << "Compute Gate Leakage: " << computeGateLeakage << " W" << std::endl;
    // std::cout << "Memory Gate Leakage: " << memoryGateLeakage << " W" << std::endl;
    std::cout << std::fixed << "Power (W): " << computeDynPower + memoryDynPower + computeSubThreshLeakage + memorySubThreshLeakage + computeGateLeakage + memoryGateLeakage << std::endl;

    std::cout << std::fixed << "Instruction Count: " << program.instructionCount << std::endl;
    // Note: PIPELINE_WIDTH - another duplicate
    const size_t PIPELINE_WIDTH = 3;
    std::cout << std::fixed << "Performance (us): " << (!program.isPipelining ?
    program.instructionCount * (1 / (CLOCK_FREQUENCY / 4)) * MICROSECONDS_PER_SECOND :
    (program.instructionCount + PIPELINE_WIDTH - 1) * (1 / CLOCK_FREQUENCY) * MICROSECONDS_PER_SECOND) << std::endl;
    std::cout << std::fixed << "Utilization: " << utilization(program) << std::endl;
    std::cout << std::fixed << "Memory Usage: " << memoryUsage(program) << " bits" << std::endl;
    */

    free(image);
    free(processed_image);
    delete [] program.instructions;
}


std::vector<std::vector<std::vector<std::vector<bool>>>> getExpectedImageForOneBitEdgeDetection(const char *imageFilename, size_t num_bits, size_t dimension, size_t expected_program_num_outputs) {
    uint8_t* image = transform_image(imageFilename, dimension, num_bits);
    std::vector<std::vector<std::vector<std::vector<bool>>>> expected_image(1, std::vector<std::vector<std::vector<bool>>>(dimension, std::vector<std::vector<bool>>(dimension, std::vector<bool>(expected_program_num_outputs, 0))));
    for (int i = 0; i < dimension; i++) {
        for (int j = 0; j < dimension; j++) {
            size_t val = image[i * dimension + j];
            expected_image[0][i][j][0] =
            (((i - 1 < 0) ? 0 : image[(i - 1) * dimension + j]) != val)
            || (((i + 1 >= dimension) ? 0 : image[(i + 1) * dimension + j]) != val)
            || (((j - 1 < 0) ? 0 : image[i * dimension + (j - 1)]) != val)
            || (((j + 1 >= dimension) ? 0 : image[i * dimension + j + 1]) != val); 
        }
    }
    free(image);
    return expected_image;
}

std::vector<std::vector<std::vector<std::vector<bool>>>> getExpectedImageForOneBitThinning(const char *imageFilename, size_t num_bits, size_t dimension, size_t expected_program_num_outputs) {
    uint8_t* image = transform_image(imageFilename, dimension, num_bits);
    std::vector<std::vector<std::vector<std::vector<bool>>>> expected_image(1, std::vector<std::vector<std::vector<bool>>>(dimension, std::vector<std::vector<bool>>(dimension, std::vector<bool>(expected_program_num_outputs, 0))));
    for (int i = 0; i < dimension; i++) {
        for (int j = 0; j < dimension; j++) {
            size_t count = 
            ((i - 1 < 0) ? 0 : image[(i - 1) * dimension + j])
            + ((i + 1 >= dimension) ? 0 : image[(i + 1) * dimension + j])
            + ((j - 1 < 0) ? 0 : image[i * dimension + j - 1])
            + ((j + 1 >= dimension) ? 0 : image[i * dimension + j + 1]);
            expected_image[0][i][j][0] = (count == 1 || count == 2) ? 0 : image[i * dimension + j];
        }
    }
    free(image);
    return expected_image;
}

std::vector<std::vector<std::vector<std::vector<bool>>>> getExpectedImageForOneBitSmoothing(const char *imageFilename, size_t num_bits, size_t dimension, size_t expected_program_num_outputs) {
    uint8_t* image = transform_image(imageFilename, dimension, num_bits);
    std::vector<std::vector<std::vector<std::vector<bool>>>> expected_image(1, std::vector<std::vector<std::vector<bool>>>(dimension, std::vector<std::vector<bool>>(dimension, std::vector<bool>(expected_program_num_outputs, 0))));
    for (int i = 0; i < dimension; i++) {
        for (int j = 0; j < dimension; j++) {
            size_t count = 
            ((i - 1 < 0) ? 0 : image[(i - 1) * dimension + j])
            + ((i + 1 >= dimension) ? 0 : image[(i + 1) * dimension + j])
            + ((j - 1 < 0) ? 0 : image[i * dimension + j - 1])
            + ((j + 1 >= dimension) ? 0 : image[i * dimension + j + 1])
            + image[i * dimension + j];
            expected_image[0][i][j][0] = count >= 3;
        }
    }
    free(image);
    return expected_image;
}

std::vector<std::vector<std::vector<std::vector<bool>>>> getExpectedImageForPrewittEdgeDetection(const char *imageFilename, size_t num_bits, size_t dimension, size_t expected_program_num_outputs) {
    uint8_t* image = transform_image(imageFilename, dimension, num_bits);
    std::vector<std::vector<std::vector<std::vector<bool>>>> expected_image(1, std::vector<std::vector<std::vector<bool>>>(dimension, std::vector<std::vector<bool>>(dimension, std::vector<bool>(expected_program_num_outputs, 0))));
    for (int i = 0; i < dimension; i++) {
        for (int j = 0; j < dimension; j++) {
            // Prewitt edge detection
            int16_t gx = ((j - 1 < 0) ? 0 : (int16_t) image[i * dimension + j - 1])
            + ((j - 1 < 0 || i + 1 >= dimension) ? 0 : (int16_t) image[(i + 1) * dimension + j - 1])
            + ((j - 1 < 0 || i - 1 < 0) ? 0 : (int16_t) image[(i - 1) * dimension + j - 1])
            - ((j + 1 >= dimension) ? 0 : (int16_t) image[i * dimension + j + 1])
            - (((j + 1 >= dimension || i + 1 >= dimension) ? 0 : (int16_t) image[(i + 1) * dimension + j + 1]))
            - (((j + 1 >= dimension || i - 1 < 0) ? 0 : (int16_t) image[(i - 1) * dimension + j + 1]));
            
            for (size_t k = 0; k < expected_program_num_outputs; k++) {
                expected_image[0][i][j][k] = (gx & (1 << k)) >> k;
            }
        }
    }

    // Print expected image
    // std::cout << "Expected image:" << std::endl;
    // for (size_t y = 0; y < dimension; y++) {
    //     for (size_t x = 0; x < dimension; x++) {
    //         uint16_t val = 0;
    //         for (size_t i = 0; i < expected_program_num_outputs; i++) {
    //             val |= expected_image[y][x][i] << i;
    //         }
    //         int16_t result = (val & 0x100) ? (int16_t) (val | 0xFE00) : (int16_t) val;
    //         std::cout << result << " ";
    //     }
    //     std::cout << std::endl;
    // }
    free(image);
    return expected_image;
}

std::vector<std::vector<std::vector<std::vector<bool>>>> getExpectedImageForMultiBitSmoothing(const char *imageFilename, size_t num_bits, size_t dimension, size_t expected_program_num_outputs) {
    uint8_t* image = transform_image(imageFilename, dimension, num_bits);
    std::vector<std::vector<std::vector<std::vector<bool>>>> expected_image(1, std::vector<std::vector<std::vector<bool>>>(dimension, std::vector<std::vector<bool>>(dimension, std::vector<bool>(expected_program_num_outputs, 0))));
    for (int i = 0; i < dimension; i++) {
        for (int j = 0; j < dimension; j++) {
            uint16_t result = (((j - 1 < 0) ? 0 : (uint16_t) image[i * dimension + j - 1])
            + ((i + 1 >= dimension) ? 0 : (uint16_t) image[(i + 1) * dimension + j])
            + ((i - 1 < 0) ? 0 : (uint16_t) image[(i - 1) * dimension + j])
            + ((j + 1 >= dimension) ? 0 : (uint16_t) image[i * dimension + j + 1])) / 4;
            
            for (size_t k = 0; k < expected_program_num_outputs; k++) {
                expected_image[0][i][j][k] = (result & (1 << k)) >> k;
            }
        }
    }
    free(image);
    return expected_image;
}

std::vector<std::vector<std::vector<std::vector<bool>>>> getExpectedImageForBinaryBPIsingModel(const char *imageFilename, size_t num_bits, size_t dimension, size_t expected_program_num_outputs, size_t num_iterations) {
    uint8_t* image = transform_image(imageFilename, dimension, num_bits);
    std::vector<std::vector<std::vector<std::vector<bool>>>> expected_images(num_iterations, std::vector<std::vector<std::vector<bool>>>(dimension, std::vector<std::vector<bool>>(dimension, std::vector<bool>(expected_program_num_outputs, 0))));
    for (size_t iter = 1; iter < num_iterations; iter++) {
        for (int i = 0; i < dimension; i++) {
            for (int j = 0; j < dimension; j++) {
                expected_images[iter][i][j][0] = 1;
            }
        }
    }
    free(image);
    return expected_images;
}

std::pair<double, double> testAllPrograms(const char *imageFilename, size_t dimension, bool useGPU) {

    uint8_t* image = transform_image(imageFilename, dimension, 1);

    // Print image
    // std::cout << "Image:" << std::endl;
    // for (size_t y = 0; y < dimension; y++) {
    //     for (size_t x = 0; x < dimension; x++) {
    //         std::cout << (int) image[y * dimension + x] << " ";
    //     }
    //     std::cout << std::endl;
    // }

    // for (size_t y = 0; y < dimension; y++) {
    //     for (size_t x = 0; x < dimension; x++) {
    //         std::cout << "offset " << y * dimension + x << ": " << (int) image[y * dimension + x] << std::endl;
    //     }
    // }

    size_t min_vliw_width = 1;
    size_t max_vliw_width = 4;
    bool do_pipelining = true;
    // Note: Need to change this if we need to add more tests
    std::vector<float> real_time_timings;
    std::vector<float> per_frame_timings;
    for (size_t vliwWidth = min_vliw_width; vliwWidth <= max_vliw_width; vliwWidth++) {
        // Note: only make pipelining tests for vliwWidth == 1
        for (size_t pipelining = 0; (pipelining <= do_pipelining && vliwWidth == 1) || pipelining == 0; pipelining++) {
            std::string directory_name = pipelining == 0 ? std::to_string(vliwWidth) + "_vliw_slot/" : "pipelining/";
            bool is_pipelining = pipelining == 1;
            testProgram(
                ("programs/" + directory_name + "edge_detection_one_bit.vis").c_str(),
                vliwWidth,
                is_pipelining,
                imageFilename,
                dimension,
                1,
                1,
                1,
                getExpectedImageForOneBitEdgeDetection(imageFilename, 1, dimension, 1),
                real_time_timings,
                per_frame_timings,
                useGPU,
                true,
                false
            );

            testProgram(
                ("programs/" + directory_name + "thinning_one_bit.vis").c_str(),
                vliwWidth,
                is_pipelining,
                imageFilename,
                dimension,
                1,
                1,
                1,
                getExpectedImageForOneBitThinning(imageFilename, 1, dimension, 1),
                real_time_timings,
                per_frame_timings,
                useGPU,
                true,
                false
            );

            testProgram(
                ("programs/" + directory_name + "smoothing_one_bit.vis").c_str(),
                vliwWidth,
                is_pipelining,
                imageFilename,
                dimension,
                1,
                1,
                1,
                getExpectedImageForOneBitSmoothing(imageFilename, 1, dimension, 1),
                real_time_timings,
                per_frame_timings,
                useGPU,
                true,
                false
            );

            testProgram(
                ("programs/" + directory_name + "prewitt_edge_detection_six_bits.vis").c_str(),
                vliwWidth,
                is_pipelining,
                imageFilename,
                dimension,
                6,
                9,
                1,
                getExpectedImageForPrewittEdgeDetection(imageFilename, 6, dimension, 9),
                real_time_timings,
                per_frame_timings,
                useGPU,
                true,
                true
            );

            testProgram(
                ("programs/" + directory_name + "smoothing_six_bits.vis").c_str(),
                vliwWidth,
                is_pipelining,
                imageFilename,
                dimension,
                6,
                6,
                1,
                getExpectedImageForMultiBitSmoothing(imageFilename, 6, dimension, 6),
                real_time_timings,
                per_frame_timings,
                useGPU,
                true,
                false
            );

            if (vliwWidth == 1 && !pipelining) {
                const size_t NUM_BP_ITERATIONS = 10;
                testProgram(
                    ("programs/" + directory_name + "binary_bp_ising_model.vis").c_str(),
                    vliwWidth,
                    is_pipelining,
                    imageFilename,
                    dimension,
                    8,
                    1,
                    NUM_BP_ITERATIONS,
                    getExpectedImageForBinaryBPIsingModel(imageFilename, 8, dimension, 1, NUM_BP_ITERATIONS),
                    real_time_timings,
                    per_frame_timings,
                    useGPU,
                    false,
                    false
                );
            }
        }
    }

    free(image);
    
    // Compute average processing time and average frame rate
    double total_real_time_duration = 0;
    double total_per_frame_duration = 0;
    size_t num_total_tests = real_time_timings.size();
    for (size_t i = 0; i < num_total_tests; i++) {
        total_real_time_duration += (double) real_time_timings[i];
        total_per_frame_duration += (double) per_frame_timings[i];
    }
    return {total_real_time_duration / ((double) num_total_tests), total_per_frame_duration / ((double) num_total_tests)};
}

int main() {
    queryGPUProperties();

    // Performance evaluation
    const char *imageFilename = "images/peacock_feather_4096.jpg";
    // for (size_t dimension = 100; dimension <= 2000; dimension += 100) {
    //     std::cout << dimension << ", ";
    //     std::pair<double, double> cpu_tests_result = testAllPrograms(imageFilename, dimension, false);
    //     std::cout << 1000.0f / cpu_tests_result.second << std::endl;
    //     std::pair<double, double> gpu_tests_result = testAllPrograms(imageFilename, dimension, true);
    //     std::cout << 1000.0f / gpu_tests_result.second << ", ";
    // }

    size_t dimension = 1000;
    // std::pair<double, double> cpu_tests_result = testAllPrograms(imageFilename, dimension, false);
    // std::cout << "Average real-time processing time (CPU): " << cpu_tests_result.first << " ms" << std::endl;
    // std::cout << "Average real-time frame rate (CPU): " << 1000.0f / cpu_tests_result.first << " fps" << std::endl;
    // std::cout << "Average per-frame processing time (CPU): " << cpu_tests_result.second << " ms" << std::endl;
    // std::cout << "Average per-frame frame rate (CPU): " << 1000.0f / cpu_tests_result.second << " fps" << std::endl;

    std::pair<double, double> gpu_tests_result = testAllPrograms(imageFilename, dimension, true);
    std::cout << "Average real-time processing time (GPU): " << gpu_tests_result.first << " ms" << std::endl;
    std::cout << "Average real-time frame rate (GPU): " << 1000.0f / gpu_tests_result.first << " fps" << std::endl;
    std::cout << "Average per-frame processing time (GPU): " << gpu_tests_result.second << " ms" << std::endl;
    std::cout << "Average per-frame frame rate (GPU): " << 1000.0f / gpu_tests_result.second << " fps" << std::endl;


    return EXIT_SUCCESS;
}